#include <stdio.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;
long int array[100];
int N;
__device__ 
/*Newton method to caculate zero point y=x*x-a */
double mysqrt(int s){
    //double s=1.0;
    double before=1.0,after;
    while(1){
        after=(before*before+s)/(2.0*before); 
        if(fabs(before-after)<0.0000001) break;
         before=after;
        
    }
    return before;
    
}
__device__ 
double  power1(int n)
{
    double temp=1.0;
    double a5=mysqrt(5);
    for(int i=0; i<n; ++i)
        temp=temp*(1.0+a5)/2.0;//sqrt

    return temp;
}
__device__ 
double  power2(int n)
{
    double temp=1.0;
    double a5=mysqrt(5);
    for(int i=0; i<n; ++i)
        temp=temp*(1.0-a5)/2.0;//sqrt

    return temp;
}


__global__ void fib(long int *C, int N)
{
    //Grid中x方向上的索引
    //int xIndex = threadIdx.x + blockIdx.x * blockDim.x;

    //Grid中y方向上的索引
    //int yIndex = threadIdx.y + blockIdx.y * blockDim.y;
    //cout<<"threadIdx.x"=<<threadIdx.x<<" xIndex="<<xIndex<<endl;

    int i = (blockIdx.x * gridDim.x + blockIdx.y) * blockDim.x * blockDim.y + threadIdx.x * blockDim.x + threadIdx.y;
    if (i < N)
    {
        C[i] = (long int)((power1(i+1)-power2(i+1))/mysqrt(5));//sqrt
    }

}



int main(int argc, char **agrv)
{

    long int *device_C;

    scanf("%d",&N);
    // N=atoi(agrv[1]);

    struct timeval tpstart,tpend;
    struct timeval kernel_begin,kernel_end;
    //开始计时
    gettimeofday(&tpstart,NULL);


    //在GPU中开辟内存
    hipMalloc((void**)&device_C, N * sizeof(long int));

    //定义block中thread的分布

    dim3 threadsPerBlock(4, 4);
    //根据输入图片的宽高定义block的大小
    dim3 blocksPerGrid(1,(N + 15) / 16,1);

    gettimeofday(&kernel_begin,NULL);
    fib <<<blocksPerGrid, threadsPerBlock >>>(device_C,N);
    gettimeofday(&kernel_end,NULL);
    double kenerl_time=1000000*(kernel_end.tv_sec-kernel_begin.tv_sec)+kernel_end.tv_usec-kernel_begin.tv_usec;
    cout<<"this kernel operation consumes "<<kenerl_time<<"us\n";


    //将结果传回CPU
    hipMemcpy(array, device_C, N * sizeof(long int), hipMemcpyDeviceToHost);
    for(int i = 0 ; i<N ; i++)
    {
        if(i==0) printf("%ld",array[i]);
        else printf(" %ld",array[i]);
    }
    putchar('\n');

    //结束计时
    gettimeofday(&tpend,NULL);
    double timeuse=1000000*(tpend.tv_sec-tpstart.tv_sec)+tpend.tv_usec-tpstart.tv_usec;
    cout<<"this operation consumes "<<timeuse/1000<<"ms\n";

    return 0;
}


/***** End *****/

